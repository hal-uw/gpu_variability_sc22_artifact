#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include <cmath>
#include "gputimer.h"
#include <string>
#include <sys/stat.h>
#include <fstream>
#include <sstream>

// Function to read data from file
float* read_from_file(std::string file_name){
  // lets get filesize
  	struct stat results;
  	if (stat(file_name.c_str(), &results) != 0){
		// An error occurred
		std::cout << "ERROR: unable to get filesize" << std::endl;
		return NULL;
	}
	// The size of the file in bytes is in results.st_size
	// Lets allocate an array to contain the binary file
	float* data = (float *)malloc(results.st_size);

	// lets write it to binary file
	std::ifstream infile;

	// open a binary file
	infile.open(file_name.c_str(), std::fstream::binary | std::fstream::in);

	//read data from file
	infile.read((char*) data, results.st_size);

	// close the file
	infile.close();

	return data;
}



// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Randomization helpers
// adapted from https://github.com/ROCmSoftwarePlatform/rocBLAS/blob/rocm-3.0/clients/include/rocblas_init.hpp#L42

void fill_sin(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(float(i + j * nr_rows_A));
}


void fill_cos(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = cos(float(i + j * nr_rows_A));
}

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul( hipblasHandle_t handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
  	hipblasStatus_t err = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
  	if (err != HIPBLAS_STATUS_SUCCESS)
  		std::cout << "Error: " <<  _cudaGetErrorEnum(err) << std::endl;

}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int def(int value, int reps, int device) {

	hipSetDevice(device);
  	hipStream_t computeStream;
  	hipError_t result;
  	result = hipStreamCreate(&computeStream);

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = value;

  	// float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	// float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

	// If you already have useful values in A and B you can copy them in GPU:
	// hipMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice);
	// hipMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice);

	// Fill the arrays A and B on GPU with random numbers
	// GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	// GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);
	// fill_sin(h_A, nr_rows_A, nr_cols_A);
	// fill_cos(h_B, nr_rows_B, nr_cols_B);

	// Adding ability to read the data generated directly from disk to reduce
	// gpu utilizaion between multiple runs
	std::stringstream ss;
	ss << value;
	std::string fname_A = std::string("host_A_") + ss.str() + std::string(".bin");
	std::string fname_B = std::string("host_B_") + ss.str() + std::string(".bin");
	
	float *h_A;
	float *h_B;
	h_A = read_from_file(fname_A);
	h_B = read_from_file(fname_B);

	// Declare a new time
	GpuTimer memcpy_timer;
	memcpy_timer.Start();

	// Optionally we can copy the data back on CPU and print the arrays
	hipMemcpyAsync(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, computeStream);
	hipMemcpyAsync(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, computeStream);
	std::cout << "A =" << std::endl;
	// print_matrix(h_A, nr_rows_A, nr_cols_A);
	std::cout << "B =" << std::endl;
  	// print_matrix(h_B, nr_rows_B, nr_cols_B);

	// Ensure memcpy completes before kernel launches
	cudaStreamSynchronisze(computeStream);
	memcpy_timer.Stop();
	std::cout <<"Memcpy Runtime (ms) = " << memcpy_timer.Elapsed() << std::endl;

  	// Create a handle for CUBLAS
	hipblasHandle_t handle;
  	hipblasCreate(&handle);
  	hipblasSetStream(handle, computeStream);
  	GpuTimer timer;

  	for (int i=0; i< reps; i++){
		// Multiply A and B on GPU
		timer.Start();
  		gpu_blas_mmul(handle, d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
  		hipStreamSynchronize(computeStream);
        timer.Stop();
		std::cout <<"Kernel " << i << " Runtime = " << timer.Elapsed() << std::endl;
  	}

	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpyAsync(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	std::cout << "C =" << std::endl;
	// print_matrix(h_C, nr_rows_C, nr_cols_C);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
    hipFree(d_C);

  	result = hipStreamDestroy(computeStream);

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}

int main(int argc, char* argv[]){
	// for (int i=100; i <= 100000; i = i*10){
	// 	std::cout << "\n\n\n" << i << "\n";
	// 	def(1024, i);
	// }
	if (argc != 4){
		std::cout << "Usage: mul <dim> <reps> <target-device num>" << std::endl;
		exit(-1);
	}
	int dim = atoi(argv[1]);
	int reps = atoi(argv[2]);
	int device = atoi(argv[3]);
	//cout << dim <<
	def(dim, reps, device);
	return 0;
}
