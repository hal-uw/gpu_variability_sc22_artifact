#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <nppi.h>
#include <stdio.h>

#define INF (255.0f * 255.0f * 3 * 8 + 1)
#define _FIXED(x) rintf(1e1f * (x))

struct {
	float det;
	float sigma_inv[9];
	unsigned int count;
} GMM_t;


__device__
__forceinline__
float get_component(uchar4 pixel, int i) {
	switch(i) {		
		case 0 :
			return 1.0f;
		case 1 :
			return pixel.x;
		case 2 :
			return pixel.y;	
		case 3 :
			return pixel.z;
		case 4 :
			return pixel.x * pixel.x;
		case 5 :
			return pixel.x * pixel.y;
		case 6 :
			return pixel.x * pixel.z;
		case 7 :
			return pixel.y * pixel.y;
		case 8 :
			return pixel.y * pixel.z;
		case 9 :
			return pixel.z * pixel.z;
	};

	return 0.0f;
}

__device__
__forceinline__
float get_constant(float* gmm, int i) {
	const float epsilon = 1.0e-3f;

	switch(i) {		
		case 0 :
			return 0.0f;
		case 1 :
			return 0.0f;
		case 2 :
			return 0.0f;	
		case 3 :
			return 0.0f;
		case 4 :
			return gmm[1] * gmm[1] + epsilon;
		case 5 :
			return gmm[1] * gmm[2];
		case 6 :
			return gmm[1] * gmm[3];
		case 7 :
			return gmm[2] * gmm[2] + epsilon;
		case 8 :
			return gmm[2] * gmm[3];
		case 9 :
			return gmm[3] * gmm[3] + epsilon;
	};

	return 0.0f;
}


// Tile Size: 32x32, Block Size 32xwarp_N
template<int warp_N, bool create_gmm_flags>
__global__
void GMMReductionKernel(int gmm_idx, float* gmm, int gmm_pitch, const uchar4* image, int image_pitch, unsigned char* alpha, int alpha_pitch, int width, int height, unsigned int* tile_gmms)
{
	__shared__ uchar4 s_lists[32*32];
	__shared__ volatile float s_gmm[32*warp_N];
	__shared__ float s_final[warp_N];

	__shared__ int gmm_flags[32];

	const int warp_idx = threadIdx.y;
	const int thread_idx = threadIdx.y * 32 + threadIdx.x;
	const int lane_idx = threadIdx.x;

	float* block_gmm = &gmm[(gridDim.x * gridDim.y * gmm_idx + blockIdx.y * gridDim.x + blockIdx.x) * gmm_pitch];
	volatile float* warp_gmm = &s_gmm[warp_idx * 32];

	if( create_gmm_flags ) {		
		if( threadIdx.y == 0 ) gmm_flags[threadIdx.x] = 0;
		__syncthreads();
	} else {
		unsigned int gmm_mask = tile_gmms[blockIdx.y * gridDim.x + blockIdx.x];

		if( (gmm_mask & (1u << gmm_idx)) == 0 ) {

			if( threadIdx.x < 10 && threadIdx.y ==0 ) {
				block_gmm[threadIdx.x] = 0.0f;
			}

			return;
		}
	}

	int list_idx = 0;

	int y = blockIdx.y * 32 + threadIdx.y;
	int x = blockIdx.x * 32 + threadIdx.x;


	// Build lists of pixels that belong to this GMM

	for( int k=0; k < (32/warp_N); ++k) { 
		if( x < width && y < height ) {
			int my_gmm_idx = alpha[y * alpha_pitch + x];

			if( create_gmm_flags )
				gmm_flags[my_gmm_idx] = 1;

			if( my_gmm_idx == gmm_idx ) {
				uchar4 pixel = image[y * image_pitch + x];
				s_lists[thread_idx + list_idx * (32*warp_N)] = pixel;
				++list_idx;
			}
		}

		y += warp_N;
	}

	__syncthreads();

	if( threadIdx.y == 0 && create_gmm_flags ) {
#if __CUDA_ARCH__ < 200	
		unsigned int gmm_flags_bvec = 0;
		for( int i=0;i<32;++i) {
			if( gmm_flags[i] > 0 ) gmm_flags_bvec |= 1 << i;
		}

		tile_gmms[blockIdx.y * gridDim.x + blockIdx.x] = gmm_flags_bvec;		
#else		
		tile_gmms[blockIdx.y * gridDim.x + blockIdx.x] = __ballot(gmm_flags[threadIdx.x] > 0);
#endif
	}

	// Reduce for each global GMM element

	for( int i=0; i<10; ++i ) {
		float thread_gmm;

		if( i == 0 ) {
			// thread_gmm = list_idx for first component
			thread_gmm = list_idx;
		} else {
			thread_gmm = list_idx > 0 ? get_component(s_lists[thread_idx],i) : 0.0f;

			for( int k=1; k<(32/warp_N) && k < list_idx; ++k ) {
				thread_gmm += get_component(s_lists[thread_idx + k * (32*warp_N)], i);	
			}
		}

		warp_gmm[lane_idx] = thread_gmm;

		// Warp Reductions
		thread_gmm += warp_gmm[(lane_idx + 16) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 8) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 4) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 2) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 1) & 31];		
		s_final[warp_idx] = thread_gmm;

		__syncthreads();

		// Final Reduction
		if( warp_idx ==0 && lane_idx == 0 ) {	
			for( int j=1; j<warp_N; ++j)
				thread_gmm += s_final[j];

			block_gmm[i] = thread_gmm;
		}
	}	

}

__constant__ int det_indices[] = { (9 << (4*4)) + (4 << (3*4)) + (6 << (2*4)) + (5 << (1*4)) + (4 << (0*4)),
(5 << (4*4)) + (8 << (3*4)) + (6 << (2*4)) + (6 << (1*4)) + (7 << (0*4)),
(5 << (4*4)) + (8 << (3*4)) + (7 << (2*4)) + (8 << (1*4)) + (9 << (0*4))
};

__constant__ int inv_indices[] = { (4 << (5*4)) + (5 << (4*4)) + (4 << (3*4)) + (5 << (2*4)) + (6 << (1*4)) + (7 << (0*4)),
(7 << (5*4)) + (6 << (4*4)) + (9 << (3*4)) + (8 << (2*4)) + (8 << (1*4)) + (9 << (0*4)),
(5 << (5*4)) + (4 << (4*4)) + (6 << (3*4)) + (6 << (2*4)) + (5 << (1*4)) + (8 << (0*4)),
(5 << (5*4)) + (8 << (4*4)) + (6 << (3*4)) + (7 << (2*4)) + (9 << (1*4)) + (8 << (0*4))
};


// One block per GMM, 32*warp_N threads (1-dim)
template <int warp_N, bool invertSigma>
__global__
void GMMFinalizeKernel(float* gmm, float* gmm_scratch, int gmm_pitch, int N)
{
	__shared__ volatile float s_gmm[warp_N*32];
	__shared__ float s_final[warp_N];
	__shared__ float final_gmm[15];

	const int thread_N = warp_N * 32;

	float* gmm_partial = &gmm_scratch[N*blockIdx.x*gmm_pitch];

	volatile float* warp_gmm = &s_gmm[threadIdx.x & 0x0ffe0];

	int thread_idx = threadIdx.x;
	int lane_idx = threadIdx.x & 31;
	int warp_idx = threadIdx.x >> 5;

	float norm_factor = 1.0f;

	for( int i=0; i<10; ++i ) {
		float thread_gmm = 0.0f;

		for( int j=thread_idx; j < N; j+= thread_N ) {
			thread_gmm += gmm_partial[j * gmm_pitch + i];
		}

		warp_gmm[lane_idx] = thread_gmm;

		// Warp Reduction
		thread_gmm += warp_gmm[(lane_idx + 16) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 8) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 4) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 2) & 31];
		warp_gmm[lane_idx] = thread_gmm;

		thread_gmm += warp_gmm[(lane_idx + 1) & 31];		

		s_final[warp_idx] = thread_gmm;

		__syncthreads();

		// Final Reduction
		if( warp_idx ==0 && lane_idx == 0 ) {	
			for( int j=1; j<warp_N; ++j)
				thread_gmm += s_final[j];

			final_gmm[i] = norm_factor * thread_gmm - get_constant(final_gmm, i);			

			if( i == 0 ) {				
				if( thread_gmm > 0 )
					norm_factor = 1.0f / thread_gmm;	
			}
		}		
	}

	if( threadIdx.y == 0 ) {

		// Compute det(Sigma) using final_gmm [10-14] as scratch mem

		if( threadIdx.x < 5 ) {

			int idx0 = (det_indices[0] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
			int idx1 = (det_indices[1] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
			int idx2 = (det_indices[2] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);

			final_gmm[10 + threadIdx.x] = final_gmm[idx0] * final_gmm[idx1] * final_gmm[idx2];

			float det = final_gmm[10] + 2.0f * final_gmm[11] - final_gmm[12] - final_gmm[13] - final_gmm[14];
			final_gmm[10] = det;
		}

		// Compute inv(Sigma)
		if( invertSigma && threadIdx.x < 6 ) {
			int idx0 = (inv_indices[0] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
			int idx1 = (inv_indices[1] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
			int idx2 = (inv_indices[2] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
			int idx3 = (inv_indices[3] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);

			float temp = final_gmm[idx0] * final_gmm[idx1] - final_gmm[idx2] * final_gmm[idx3];

			if( final_gmm[10] > 0.0f ) {
				final_gmm[4+threadIdx.x] = temp / final_gmm[10];
			} else {
				final_gmm[4+threadIdx.x] = 0.0f;
			}
		}

		if( threadIdx.x < 11 )
			gmm[blockIdx.x * gmm_pitch + threadIdx.x] = final_gmm[threadIdx.x];

	}
}


// Single block, 32x2
__global__
void GMMcommonTerm( int gmmK, float* gmm, int gmm_pitch) {

	__shared__ volatile float s_n[2][32];

	int gmm_idx = (threadIdx.x * 2) | threadIdx.y;

	float gmm_n = threadIdx.x < gmmK ? gmm[gmm_idx * gmm_pitch] : 0.0f;
	float sum = gmm_n;
	s_n[threadIdx.y][threadIdx.x] = sum;

	// Warp Reduction
	sum += s_n[threadIdx.y][(threadIdx.x + 16) & 31];
	s_n[threadIdx.y][threadIdx.x] = sum;

	sum += s_n[threadIdx.y][(threadIdx.x + 8) & 31];
	s_n[threadIdx.y][threadIdx.x] = sum;

	sum += s_n[threadIdx.y][(threadIdx.x + 4) & 31];
	s_n[threadIdx.y][threadIdx.x] = sum;

	sum += s_n[threadIdx.y][(threadIdx.x + 2) & 31];
	s_n[threadIdx.y][threadIdx.x] = sum;

	sum += s_n[threadIdx.y][(threadIdx.x + 1) & 31];

	if( threadIdx.x < gmmK ) {
		float det = gmm[gmm_idx * gmm_pitch + 10];
		float commonTerm =  gmm_n / (sqrtf(det) * sum); 

		gmm[gmm_idx * gmm_pitch + 10] = commonTerm;
	}
}

hipError_t GMMUpdate(int gmm_N, float* gmm, float* scratch_mem, int gmm_pitch, const uchar4* image, int image_pitch, unsigned char* alpha, int alpha_pitch, int width, int height) {
	dim3 grid( (width+31) / 32, (height+31) / 32);
	dim3 block(32,4);


	GMMReductionKernel<4, true><<<grid, block>>>(0, &scratch_mem[grid.x * grid.y], gmm_pitch/4, image, image_pitch/4, alpha, alpha_pitch, width, height, (unsigned int*) scratch_mem);
	for( int i=1; i<gmm_N; ++i) {
		GMMReductionKernel<4, false><<<grid, block>>>(i, &scratch_mem[grid.x * grid.y], gmm_pitch/4, image, image_pitch/4, alpha, alpha_pitch, width, height, (unsigned int*) scratch_mem);
	}

	GMMFinalizeKernel<4, true><<<gmm_N, 32*4>>>(gmm, &scratch_mem[grid.x * grid.y], gmm_pitch/4, grid.x * grid.y);

	block.x = 32;block.y = 2;
	GMMcommonTerm<<<1, block>>>(gmm_N / 2, gmm, gmm_pitch/4);

	return hipGetLastError();
}



__device__
float GMMTerm(uchar4 pixel, const float* gmm)
{
	float3 v = make_float3(pixel.x - gmm[1], pixel.y - gmm[2], pixel.z - gmm[3]);

	float xxa = v.x * v.x * gmm[4];
	float yyd = v.y * v.y * gmm[7];
	float zzf = v.z * v.z * gmm[9];

	float yxb = v.x * v.y * gmm[5];
	float zxc = v.z * v.x * gmm[6];
	float zye = v.z * v.y * gmm[8];

	return gmm[10] * expf( -0.5f * (xxa + yyd + zzf + 2.0f * (yxb + zxc + zye)) );
}

__global__
void DataTermKernel(Npp32s* terminals, int terminal_pitch, int gmmN, const float* gmm, int gmm_pitch, const uchar4* image, int image_pitch, const unsigned char* trimap, int trimap_pitch, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < width && y < height ) {
		unsigned char c = trimap[y*trimap_pitch+x];

		Npp32f data;

		if( c == 0 ) {
			// Definitely Background
			data = -INF;
		} else if( c == 2 ) {
			// Definitely Foreground
			data = + INF;
		} else {
			// Unknown
			uchar4 pixel = image[y * image_pitch + x];

			Npp32f data_bg = GMMTerm(pixel, gmm);
			Npp32f data_fg = GMMTerm(pixel, &gmm[gmm_pitch]);

			for( int i=2; i<gmmN; i+=2 ) {		
				data_bg += GMMTerm(pixel, &gmm[(i) * gmm_pitch]);
				data_fg += GMMTerm(pixel, &gmm[(i+1) * gmm_pitch]);
			}

			data_bg = -logf(data_bg);
			data_fg = -logf(data_fg);

			data = data_bg - data_fg;
			data = max(min(data, INF),-INF);
		}

		terminals[y*terminal_pitch + x] = _FIXED(data);
	}
}


hipError_t DataTerm(Npp32s* terminals, int terminal_pitch, int gmmN, const float* gmm, int gmm_pitch, const uchar4* image, int image_pitch, const unsigned char* trimap, int trimap_pitch, int width, int height) {

	dim3 block(32,8);
	dim3 grid( (width+block.x-1) / block.x, (height+block.y-1) / block.y);

	DataTermKernel<<<grid, block>>>(terminals, terminal_pitch/4, gmmN, gmm, gmm_pitch/4, image, image_pitch/4, trimap, trimap_pitch, width, height);

	return hipGetLastError();
}


__global__
void GMMAssignKernel(int gmmN, const float* gmm, int gmm_pitch, const uchar4* image, int image_pitch, unsigned char* g_alpha, int alpha_pitch, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < width && y < height ) {
		unsigned char alpha = g_alpha[y*alpha_pitch+x] & 1;

		// Unknown
		uchar4 pixel = image[y * image_pitch + x];

		int alpha_min = alpha;
		float max_prob = GMMTerm(pixel, &gmm[(alpha_min) * gmm_pitch]);

		for( int i=alpha+2; i<gmmN; i+=2 ) {		
			float prob = GMMTerm(pixel, &gmm[(i) * gmm_pitch]);
			if( prob > max_prob) {
				alpha_min = i;
				max_prob = prob;
			}
		}

		g_alpha[y*alpha_pitch+x] = alpha_min;
	}
}

hipError_t GMMAssign(int gmmN, const float* gmm, int gmm_pitch, const uchar4* image, int image_pitch, unsigned char* alpha, int alpha_pitch, int width, int height) {

	dim3 block(32,16);
	dim3 grid( (width+block.x-1) / block.x, (height+block.y-1) / block.y);

	GMMAssignKernel<<<grid, block>>>(gmmN, gmm, gmm_pitch/4, image, image_pitch/4, alpha, alpha_pitch, width, height);

	return hipGetLastError();
}

__device__
float3 normalize(float3 v)
{
	float norm = 1.0f / sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);

	return make_float3(v.x * norm, v.y * norm, v.z * norm);
}

__device__
float3 mul_right(const float* M, float3 v) {	
	return make_float3(
		M[0] * v.x + M[1] * v.y + M[2] * v.z,
		M[1] * v.x + M[3] * v.y + M[4] * v.z,
		M[2] * v.x + M[4] * v.y + M[5] * v.z);
}

__device__
float largest_eigenvalue(const float* M)
{
	float norm = M[0] > M[3] ? M[0] : M[3];
	norm = M[0] > M[5] ? M[0] : M[5];
	norm = 1.0f / norm;

	float a00 = norm * M[0];
	float a01 = norm * M[1];
	float a02 = norm * M[2];
	float a11 = norm * M[3];
	float a12 = norm * M[4];
	float a22 = norm * M[5];

	float c0 = a00*a11*a22 + 2.0f*a01*a02*a12 - a00*a12*a12 - a11*a02*a02 - a22*a01*a01;
	float c1 = a00*a11 - a01*a01 + a00*a22 - a02*a02 + a11*a22 - a12*a12;
	float c2 = a00 + a11 + a22;

	const float inv3 = 1.0f / 3.0f;
	const float root3 = sqrtf(3.0f);

	float c2Div3 = c2*inv3;
	float aDiv3 = (c1 - c2*c2Div3)*inv3;

	if (aDiv3 > 0.0f) { aDiv3 = 0.0f; }

	float mbDiv2 = 0.5f*(c0 + c2Div3*(2.0f*c2Div3*c2Div3 - c1));
	float q = mbDiv2*mbDiv2 + aDiv3*aDiv3*aDiv3;

	if (q > 0.0f) { q = 0.0f; }

	float magnitude = sqrtf(-aDiv3);
	float angle = atan2(sqrtf(-q),mbDiv2)*inv3;
	float cs = cos(angle);
	float sn = sin(angle);

	float largest_eigenvalue = c2Div3 + 2.0f*magnitude*cs;

	float eigenvalue = c2Div3 - magnitude*(cs + root3*sn);
	if( eigenvalue > largest_eigenvalue)
		largest_eigenvalue = eigenvalue;

	eigenvalue = c2Div3 - magnitude*(cs - root3*sn);
	if( eigenvalue > largest_eigenvalue)
		largest_eigenvalue = eigenvalue;

	return largest_eigenvalue / norm;
}

__device__
float3 cross_prod(float3 a, float3 b) 
{
	return make_float3( (a.y*b.z)-(a.z*b.y), (a.z*b.x)-(a.x*b.z), (a.x*b.y)-(a.y*b.x) );
}

__device__
float3 compute_eigenvector(const float* M, float eigenvalue)
{
	float3 r0 = make_float3(M[0] - eigenvalue, M[1], M[2]);
	float3 r1 = make_float3(M[2] , M[3]- eigenvalue, M[4]);

	float3 eigenvector = cross_prod(r0,r1);
	return normalize(eigenvector);
}

__device__
void largest_eigenvalue_eigenvector(const float* M, float3 &evec, float &eval)
{
	eval = largest_eigenvalue(M);
	evec = compute_eigenvector(M, eval);
}

__device__
float scalar_prod(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

struct GMMSplit_t {
	int idx;
	float threshold;
	float3 eigenvector;
};

// 1 Block, 32x2
__global__
void GMMFindSplit(GMMSplit_t* gmmSplit, int gmmK, float* gmm, int gmm_pitch)
{
	__shared__ float s_eigenvalues[2][32];

	int gmm_idx = (threadIdx.x << 1) + threadIdx.y;

	float eigenvalue = 0;
	float3 eigenvector;

	if( threadIdx.x < gmmK ) {
		largest_eigenvalue_eigenvector(&gmm[gmm_idx * gmm_pitch + 4], eigenvector, eigenvalue);
	} 


	// Warp Reduction
	float maxvalue = eigenvalue;
	s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

	maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+16) & 31]);
	s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

	maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+8) & 31]);
	s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

	maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+4) & 31]);
	s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

	maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+2) & 31]);
	s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

	maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+1) & 31]);

	if( maxvalue == eigenvalue ) {
		GMMSplit_t split;

		split.idx = threadIdx.x;
		split.threshold = scalar_prod(make_float3(gmm[gmm_idx * gmm_pitch + 1], gmm[gmm_idx * gmm_pitch + 2], gmm[gmm_idx * gmm_pitch + 3]), eigenvector);
		split.eigenvector = eigenvector;

		gmmSplit[threadIdx.y] = split;
	}
}

__global__
void GMMDoSplit(const GMMSplit_t* gmmSplit, int k, float* gmm, int gmm_pitch, const uchar4* image, int image_pitch, unsigned char* alpha, int alpha_pitch, int width, int height)
{
	__shared__ GMMSplit_t s_gmmSplit[2];

	int* s_linear = (int*) s_gmmSplit;
	int* g_linear = (int*) gmmSplit;

	if( threadIdx.y ==0 && threadIdx.x < 10 ) {
		s_linear[threadIdx.x] = g_linear[threadIdx.x];
	}

	__syncthreads();

	int x = blockIdx.x * 32 + threadIdx.x;
	int y0 = blockIdx.y * 32;

	for( int i = threadIdx.y; i < 32; i += blockDim.y ) {		
		int y = y0 + i;

		if( x < width && y < height ) {			
			unsigned char my_alpha = alpha[y * alpha_pitch + x];

			int select = my_alpha & 1;
			int gmm_idx = my_alpha >> 1;

			if( gmm_idx == s_gmmSplit[select].idx ) {
				// in the split cluster now
				uchar4 pixel = image[y * image_pitch + x];

				float value = scalar_prod(s_gmmSplit[select].eigenvector, make_float3(pixel.x, pixel.y, pixel.z));

				if( value > s_gmmSplit[select].threshold ) {
					// assign pixel to new cluster
					alpha[y * alpha_pitch + x] =  k + select;
				}
			}
		}
	}
}


hipError_t GMMInitialize(int gmm_N, float* gmm, float* scratch_mem, int gmm_pitch, const uchar4* image, int image_pitch, unsigned char* alpha, int alpha_pitch, int width, int height) {
	dim3 grid( (width+31) / 32, (height+31) / 32);
	dim3 block(32,4);
	dim3 smallblock(32,2);

	for( int k = 2; k < gmm_N; k+=2 ) {
		GMMReductionKernel<4, true><<<grid, block>>>(0, &scratch_mem[grid.x * grid.y], gmm_pitch/4, image, image_pitch/4, alpha, alpha_pitch, width, height, (unsigned int*) scratch_mem);

		for(int i=1;i < k; ++i) {
			GMMReductionKernel<4, false><<<grid, block>>>(i, &scratch_mem[grid.x * grid.y], gmm_pitch/4, image, image_pitch/4, alpha, alpha_pitch, width, height, (unsigned int*) scratch_mem);
		}
		GMMFinalizeKernel<4, false><<<k, 32*4>>>(gmm, &scratch_mem[grid.x * grid.y], gmm_pitch/4, grid.x * grid.y);

		GMMFindSplit<<<1, smallblock>>>((GMMSplit_t*) scratch_mem, k / 2, gmm, gmm_pitch/4);

		GMMDoSplit<<<grid, block>>>((GMMSplit_t*) scratch_mem, (k/2) << 1, gmm, gmm_pitch/4, image, image_pitch / 4, alpha, alpha_pitch, width, height);
	}

	return hipGetLastError();
}

template<class vec_a, class vec_b>
__device__
float vector_distance_2(vec_a a, vec_b b)
{
	return ((a.x-b.x)*(a.x-b.x)+(a.y-b.y)*(a.y-b.y)+(a.z-b.z)*(a.z-b.z));
}


texture<uchar4, 2, hipReadModeElementType> imageTex;

__global__
void MeanEdgeStrengthReductionKernel( int width, int height, float* scratch_mem)
{
	__shared__ volatile float s_sum[8][32];

	int y = blockIdx.y * 32 + threadIdx.y * 4;
	int x = blockIdx.x * 32 + threadIdx.x;

	float sum = 0.0f;

	for( int k=0; k < 4; ++k) { 
		if( (x > 0) && (y > 0) && (x < width-1) && (y < height-1) ) {

			uchar4 pixel = tex2D(imageTex, x + 0.5f ,y + 0.5f);
			float3 center = make_float3(pixel.x, pixel.y, pixel.z);

			sum += vector_distance_2(center, tex2D(imageTex, x - 0.5f ,y + 1.5f));
			sum += vector_distance_2(center, tex2D(imageTex, x + 0.5f ,y + 1.5f));			
			sum += vector_distance_2(center, tex2D(imageTex, x + 1.5f ,y + 1.5f));
			sum += vector_distance_2(center, tex2D(imageTex, x + 1.5f ,y + 0.5f));
		}
		++y;
	}

	// Reduce for each global GMM element
	s_sum[threadIdx.y][threadIdx.x] = sum;

	// Warp Reductions
	sum += s_sum[threadIdx.y][(threadIdx.x + 16) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 8) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 4) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 2) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 1) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	__syncthreads();

	// Final Reduction
	if( threadIdx.y ==0 && threadIdx.x == 0 ) {	
		for( int j=1; j<8; ++j)
			sum += s_sum[j][0];

		scratch_mem[blockIdx.y * gridDim.x + blockIdx.x] = sum / (4.0f * (width-2.0f) * (height-2.0f) );
	}
}

__global__
void MeanEdgeStrengthFinalKernel( float* scratch_mem, int N)
{
	__shared__ volatile float s_sum[4][32];


	int idx = threadIdx.y * blockDim.x + threadIdx.x;
	int N_threads = blockDim.x * blockDim.y;

	float sum = idx < N ? scratch_mem[idx] : 0.0f;

	for( idx += N_threads; idx < N; idx += N_threads ) {
		sum += scratch_mem[idx];
	}
	s_sum[threadIdx.y][threadIdx.x] = sum;

	// Warp Reduction
	sum += s_sum[threadIdx.y][(threadIdx.x + 16) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 8) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 4) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 2) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	sum += s_sum[threadIdx.y][(threadIdx.x + 1) & 31];
	s_sum[threadIdx.y][threadIdx.x] = sum;

	__syncthreads();

	if( threadIdx.y ==0 && threadIdx.x == 0 ) {	
		for( int j=1; j<4; ++j)
			sum += s_sum[j][0];

		// Store beta
		scratch_mem[0] = 1.0f/(2.0f *  sum);
	}

}

__device__
Npp32f edge_weight(float3 zm, uchar4 zn, float alpha, float beta, float recp_dist)
{	
	return recp_dist * alpha * expf(-beta * (vector_distance_2(zm, make_float3(zn.x, zn.y, zn.z)))) + 3.0f;
}


__global__
void EdgeCuesKernel( float alpha, const float* g_beta, Npp32s* g_left_transposed, Npp32s* g_right_transposed, Npp32s* g_top, Npp32s * g_bottom, Npp32s* g_topleft, Npp32s* g_topright, Npp32s* g_bottomleft, Npp32s* g_bottomright, int pitch, int transposed_pitch, int width, int height ) {

	__shared__ Npp32s s_right[32][33];

	int y0 = blockIdx.y * 32;
	int x0 = blockIdx.x * 32;

	int x = x0+threadIdx.x;

	const float beta = g_beta[0];

	for( int i=threadIdx.y; i < 32; i+=blockDim.y ) {
		int y = y0 + i;

		if( x < width && y < height ) {

			uchar4 pixel = tex2D(imageTex, x + 0.5f ,y + 0.5f);
			float3 center = make_float3(pixel.x, pixel.y, pixel.z);

			// Left/Right
			s_right[i][threadIdx.x] = edge_weight( center, tex2D(imageTex, x + 1.5f ,y + 0.5f), alpha, beta, 1.0f);

			// Top/Bottom
			Npp32s bottom;
			bottom = _FIXED( edge_weight( center, tex2D(imageTex, x + 0.5f ,y + 1.5f), alpha, beta, 1.0f) );

			if( y < height -1 ) {
				g_bottom[y * pitch + x] = bottom;
				g_top[(y+1) * pitch + x] = bottom;
			} else {
				g_bottom[y * pitch + x] = 0;
				g_top[x] = 0;
			}

			// Bottomright
			Npp32s bottomright;
			bottomright = _FIXED( edge_weight( center, tex2D(imageTex, x + 1.5f ,y + 1.5f), alpha, beta, 1.0f / sqrtf(2.0f)) );

			if( y < height-1 && x < width-1 ) {
				g_bottomright[y * pitch + x] = bottomright;
			} else {
				g_bottomright[y * pitch + x] = 0;
			}

			// Bottomleft
			Npp32s bottomleft;
			bottomleft = _FIXED( edge_weight( center, tex2D(imageTex, x - 0.5f ,y + 1.5f), alpha, beta, 1.0f / sqrtf(2.0f)));

			if( y < height-1 && x > 0 ) {
				g_bottomleft[y * pitch + x] = bottomleft;
			} else {
				g_bottomleft[y * pitch + x] = 0;
			}

			// topright
			Npp32s topright;
			topright = _FIXED( edge_weight( center, tex2D(imageTex, x + 1.5f ,y - 0.5f), alpha, beta, 1.0f / sqrtf(2.0f)) );

			if( y > 0 && x < width-1 ) {
				g_topright[y * pitch + x] = topright;
			} else {
				g_topright[y * pitch + x] = 0;
			}

			// topleft
			Npp32s topleft;
			topleft = _FIXED( edge_weight( center, tex2D(imageTex, x - 0.5f ,y - 0.5f), alpha, beta, 1.0f / sqrtf(2.0f)) );
			if( y > 0  && x > 0 ) {
				g_topleft[y * pitch + x] = topleft;
			} else {
				g_topleft[y * pitch + x] = 0;
			}

		}
	}
	__syncthreads();

	int y = y0 + threadIdx.x;

	for( int i=threadIdx.y; i < 32; i+=blockDim.y ) {

		int x = x0 + i;

		if( x < width && y < height ) {
			if( x < width - 1 ) {
				g_right_transposed[x * transposed_pitch +y] = s_right[threadIdx.x][i];
				g_left_transposed[(x+1) * transposed_pitch +y] = s_right[threadIdx.x][i];
			} else {
				g_right_transposed[x * transposed_pitch +y] = 0;
				g_left_transposed[y] = 0;
			}
		}
	}
}

hipError_t EdgeCues( float alpha, const uchar4* image, int image_pitch, Npp32s* left_transposed, Npp32s* right_transposed, Npp32s* top, Npp32s * bottom, Npp32s* topleft, Npp32s* topright, Npp32s* bottomleft, Npp32s* bottomright, int pitch, int transposed_pitch, int width, int height, float* scratch_mem  )
{
	hipError_t error;

	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc<uchar4>();

	error = hipBindTexture2D(NULL, imageTex, image, channelDesc, width, height,  image_pitch);
	if( error != hipSuccess ) return error;

	dim3 grid( (width+31) / 32, (height+31) / 32);
	dim3 block(32,4);
	dim3 large_block(32,8);

	MeanEdgeStrengthReductionKernel<<<grid, large_block>>>( width, height, scratch_mem);
	MeanEdgeStrengthFinalKernel<<<1,block>>>( scratch_mem, grid.x * grid.y);

	EdgeCuesKernel<<<grid, block>>>( alpha , scratch_mem, left_transposed, right_transposed, top, bottom, topleft, topright, bottomleft, bottomright, pitch / 4, transposed_pitch/ 4, width, height );

	error = hipUnbindTexture(imageTex);
	return error;
}


__global__
void SegmentationChangedKernel(int* g_changed, Npp8u* alpha_old, Npp8u* alpha_new, int alpha_pitch, int width, int height) 
{
#if __CUDA_ARCH__ < 200	
	__shared__ int s_changed;
	s_changed = 0;
	__syncthreads();
#endif

	int x = blockIdx.x * 32 + threadIdx.x;
	int y = blockIdx.y * 32 + threadIdx.y;

	int changed = 0;

	for( int k=0; k < 4; ++k ) {
		if( x < width && y < height ) {
			changed |= (alpha_old[y * alpha_pitch+x] ^ alpha_new[y * alpha_pitch+x]) & 1;
		}
		y += blockDim.y;
	}

		
#if __CUDA_ARCH__ < 200	
	if( changed > 0 )
		s_changed = 1;

	__syncthreads();

	if( threadIdx.y == 0 && s_changed > 0 ) {
		g_changed[0] = 1;
	}

#else
	if( __syncthreads_or(changed > 0 ) ) {
		if( threadIdx.y == 0 ) {
			g_changed[0] = 1;
		}
	}
#endif
}


hipError_t SegmentationChanged(bool &result, int* d_changed, Npp8u* alpha_old, Npp8u* alpha_new, int alpha_pitch, int width, int height) 
{
	hipError_t error;
	dim3 grid( (width+31) / 32, (height+31) / 32);
	dim3 block(32,8);

	error = hipMemsetAsync(d_changed,0,4);
	if( error != hipSuccess ) return error;

	SegmentationChangedKernel<<<grid, block>>>(d_changed, alpha_old, alpha_new, alpha_pitch, width, height);

	int h_changed;
	error = hipMemcpy(&h_changed, d_changed, 4, hipMemcpyDeviceToHost);

	result = (h_changed != 0);
	return error;
}


struct boxfilter_functor {
__device__
uchar4 operator()(const uchar4 &a, const uchar4 &b, const uchar4 &c, const uchar4 &d) 
{
	float4 r = make_float4(0.25f * a.x, 0.25f * a.y, 0.25f * a.z, 0.25f * a.w);

	r.x += 0.25f * b.x; r.y += 0.25f * b.y; r.z += 0.25f * b.z; r.w += 0.25f * b.w;
	r.x += 0.25f * c.x; r.y += 0.25f * c.y; r.z += 0.25f * c.z; r.w += 0.25f * c.w;
	r.x += 0.25f * d.x; r.y += 0.25f * d.y; r.z += 0.25f * d.z; r.w += 0.25f * d.w;
	
	return make_uchar4(rintf(r.x), rintf(r.y), rintf(r.z), rintf(r.w));
}
};

struct maxfilter_functor {
__device__
unsigned char operator()(const unsigned char &a, const unsigned char &b, const unsigned char &c, const unsigned char &d) 
{
	return max(max(max(a,b),c),d);
}
};

struct minfilter_functor {
__device__
unsigned char operator()(const unsigned char &a, const unsigned char &b, const unsigned char &c, const unsigned char &d) 
{
	return min(min(min(a,b),c),d);
}
};


template<class T>
__device__
T clamp_read(int y0, int x0, const T* image, int pitch, int width, int height)
{
	int x = min(x0, width-1);
	int y = min(y0, height-1);
	
	return image[y * pitch + x];
}

template<class T, class functor_t>
__global__
void downscaleKernel(T* small_image, int small_pitch, int small_width, int small_height, const T* image, int pitch, int width, int height, functor_t functor) {
	__shared__ T tile[16][64];

	int x0 = blockIdx.x * 64 + threadIdx.x;
	int y0 = blockIdx.y * 64 + threadIdx.y;

	int small_x0 = blockIdx.x * 32 + threadIdx.x;
	int small_y0 = blockIdx.y * 32 + threadIdx.y;


	for( int k=0; k < 4; ++k ) {
		int y = y0 + k * 16;

		tile[threadIdx.y][threadIdx.x] = clamp_read(y,x0,image,pitch, width, height);
		tile[threadIdx.y][threadIdx.x+32] =  clamp_read(y,x0+32,image,pitch, width, height);
		tile[threadIdx.y+8][threadIdx.x+32] = clamp_read(y+8,x0+32,image,pitch, width, height);
		tile[threadIdx.y+8][threadIdx.x] = clamp_read(y+8,x0,image,pitch, width, height);

		__syncthreads();

		int small_y = small_y0 + k * 8;

		if( small_y < small_height && small_x0 < small_width ) {
			small_image[small_y * small_pitch + small_x0] = functor( tile[2 * threadIdx.y][2 * threadIdx.x], 
				tile[2 * threadIdx.y][2 * threadIdx.x+1],
				tile[2 * threadIdx.y+1][2 * threadIdx.x+1],
				tile[2 * threadIdx.y+1][2 * threadIdx.x]);
		}
	}
}

hipError_t downscale(uchar4* small_image, int small_pitch, int small_width, int small_height, const uchar4* image, int pitch, int width, int height) {
	
	dim3 grid((width + 63)/64, (height+63)/64);
	dim3 block(32,8);

	downscaleKernel<<<grid, block>>>(small_image, small_pitch/4, small_width, small_height, image, pitch/4, width, height, boxfilter_functor());

	return hipGetLastError();
}

hipError_t downscaleTrimap(unsigned char* small_image, int small_pitch, int small_width, int small_height, const unsigned char* image, int pitch, int width, int height) {
	
	dim3 grid((width + 63)/64, (height+63)/64);
	dim3 block(32,8);

	downscaleKernel<<<grid, block>>>(small_image, small_pitch, small_width, small_height, image, pitch, width, height, maxfilter_functor());
	return hipGetLastError();
}


__global__
void upsampleAlphaKernel(unsigned char* alpha, unsigned char* small_alpha, int alpha_pitch, int width, int height, int shift)
{
	int x = blockIdx.x * 128 + threadIdx.x * 4;
	int y0 = blockIdx.y * 32 + threadIdx.y;

	uchar4 * alpha4 = (uchar4*) alpha;
	int alpha4_pitch = alpha_pitch / 4;

	for( int k=0; k<4; ++k ) {
		int y = y0 + k*8;
		uchar4 output;

		if( x < width && y < height ) {
			output.x = small_alpha[(y >> shift) * alpha_pitch + (x >> shift)];
			output.y = small_alpha[(y >> shift) * alpha_pitch + ((x+1) >> shift)];
			output.z = small_alpha[(y >> shift) * alpha_pitch + ((x+2) >> shift)];
			output.w = small_alpha[(y >> shift) * alpha_pitch + ((x+3) >> shift)];

			alpha4[y * alpha4_pitch + blockIdx.x * 32 + threadIdx.x] = output;
		}
	}
}

hipError_t upsampleAlpha(unsigned char* alpha, unsigned char* small_alpha, int alpha_pitch, int width, int height, int small_width, int small_height)
{
	dim3 grid((width+127)/128, (height+31)/32);
	dim3 block(32,8);

	int factor = width / small_width;
	int shift = 0;

	while(factor > (1<<shift)) shift++;

	upsampleAlphaKernel<<<grid, block>>>(alpha, small_alpha, alpha_pitch, width, height, shift);

	return hipGetLastError();
}
